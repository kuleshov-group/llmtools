#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <vector>
#include <utility>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include <cuda_pipeline.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace torch::indexing;
using namespace nvcuda;

#define FULL_MASK 0xffffffff
#define HALF_MASK 0x0000ffff

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}

__device__ static inline uint32_t add_as_half2(uint32_t x, uint32_t y) {
    uint32_t z;
    asm("add.f16x2 %0,%1,%2;" : "=r"(z) : "r"(x), "r"(y));
    return z;
}


__device__ static inline uint32_t mask_lop3(uint32_t x, uint32_t m0, uint32_t m1) {
    uint32_t y;
    asm("lop3.b32 %0, %1, %2, %3, 0xEA;" : "=r"(y) : "r"(x), "r"(m0), "r"(m1));
    return y;
    // return (x & m0) | m1;
}

#define BASE_OFFSET 0xd080d080
#define XMASK 0x00f000f0
#define WMASK 0x50085008


__global__ static void
// __launch_bounds__(1024, 1024)
decode_matvec_e8p_kernel(
    float *__restrict__ output,
    const uint2 *__restrict__ input,
    const uint2 *__restrict__ weights_compressed,
    const uint32_t *__restrict__ codebook_abs,
    int N,
    int K
) {
    int warpId = threadIdx.y;
    int laneId = threadIdx.x;

    // __shared__ float sum_scratch[16*32];

    // __shared__ uint32_t codebook_local[256*32];
    // for (int icb = warpId; icb < 256; icb += 32) {
    //     codebook_local[icb*32 + laneId] = codebook_abs[icb];
    // }
    // __syncthreads();

    __shared__ uint2 shared_weights[1024*2];

    for (int iin = blockIdx.x; iin < (N >> 4); iin += gridDim.x) {

        float z0 = 0.0;
        float z1 = 0.0;
        float z2 = 0.0;
        float z3 = 0.0;

        // int shwo = laneId + 32*warpId;

        // __pipeline_memcpy_async(shared_weights + shwo, weights_compressed + laneId + 32*warpId + 1024*0 + (K >> 1)*iin, 8);
        // __pipeline_commit();

        for (int iik = warpId; iik < (K >> 6); iik += 32) {
            // if (iik + 1 < (K >> 11)) {
            //     __pipeline_memcpy_async(shared_weights + (shwo ^ 1024), weights_compressed + laneId + 32*iik + 1024 + (K >> 1)*iin, 8);
            //     __pipeline_commit();
            //     __pipeline_wait_prior(1);
            //     shwo = shwo ^ 1024;
            // }
            // else {
            //     __pipeline_wait_prior(0);
            // }

            // uint2 w_compr = shared_weights[shwo]; // weights_compressed[laneId + 32*warpId + 1024*iik + (K >> 1)*iin];
            uint2 w_compr = weights_compressed[laneId + 32*iik + (K >> 1)*iin];
            uint32_t a = w_compr.x;
            uint32_t b = w_compr.y;

            uint32_t s = b;
            s = s ^ (s >> 4);
            s = s ^ (s >> 8);
            s = s ^ (s >> 16);
            uint32_t sb = (s & 15);
            s = b ^ sb;
            sb = sb | (sb << 16);

            uint32_t input_to_warp = ((const uint32_t*)(&input[16*iik]))[laneId];
            uint32_t shifted_laneId = (laneId & 3) << 3;

            /// BLOCK 01
            {
            uint32_t x = codebook_abs[(a >> 0) & 255];
            x = x ^ ((s & 0x11111111) * 14);

            uint32_t o = BASE_OFFSET | ((sb & 0x00010001) << 4);

            uint32_t w00 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w01 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w02 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w03 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            x = codebook_abs[(a >> 8) & 255];
            x = x ^ ((s & 0x22222222) * 7);

            o = BASE_OFFSET | ((sb & 0x00020002) << 3);
            
            uint32_t w10 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w11 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w12 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w13 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            // uint2 x_in = input[0 + (laneId & 3)*4 + 16*warpId + 16*32*iik];
            // uint32_t x_in0 = x_in.x;
            // uint32_t x_in1 = x_in.y;

            uint32_t x_in0 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 0);
            uint32_t x_in1 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 1);

            asm(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
                " { %0, %1, %2, %3 },"
                " { %4, %5, %6, %7 },"
                " { %8, %9 },"
                " { %0, %1, %2, %3 };"
                : "+f"(z0), "+f"(z1), "+f"(z2), "+f"(z3)
                : "r"(w00), "r"(w10), "r"(w01),  "r"(w11),
                  "r"(x_in0), "r"(x_in1)
            );


            // x_in = input[1 + (laneId & 3)*4 + 16*warpId + 16*32*iik];
            // x_in0 = x_in.x;
            // x_in1 = x_in.y;

            x_in0 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 2);
            x_in1 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 3);

            asm(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
                " { %0, %1, %2, %3 },"
                " { %4, %5, %6, %7 },"
                " { %8, %9 },"
                " { %0, %1, %2, %3 };"
                : "+f"(z0), "+f"(z1), "+f"(z2), "+f"(z3)
                : "r"(w02), "r"(w12), "r"(w03), "r"(w13),
                  "r"(x_in0), "r"(x_in1)
            );
            }
            /// BLOCK 23 
            {
            uint32_t x = codebook_abs[(a >> 16) & 255];
            s = s >> 2;
            x = x ^ ((s & 0x11111111) * 14);

            uint32_t o = BASE_OFFSET | ((sb & 0x00040004) << 2);
            
            uint32_t w00 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w01 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w02 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w03 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            x = codebook_abs[(a >> 24) & 255];
            x = x ^ ((s & 0x22222222) * 7);

            o = BASE_OFFSET | ((sb & 0x00080008) << 1); 

            uint32_t w10 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w11 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w12 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w13 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);


            // uint2 x_in = input[2 + (laneId & 3)*4 + 16*warpId + 16*32*iik];
            // uint32_t x_in0 = x_in.x;
            // uint32_t x_in1 = x_in.y;

            uint32_t x_in0 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 4);
            uint32_t x_in1 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 5);

            asm(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
                " { %0, %1, %2, %3 },"
                " { %4, %5, %6, %7 },"
                " { %8, %9 },"
                " { %0, %1, %2, %3 };"
                : "+f"(z0), "+f"(z1), "+f"(z2), "+f"(z3)
                : "r"(w00), "r"(w10), "r"(w01), "r"(w11),
                  "r"(x_in0), "r"(x_in1)
            );


            // x_in = input[3 + (laneId & 3)*4 + 16*warpId + 16*32*iik];
            // x_in0 = x_in.x;
            // x_in1 = x_in.y;

            x_in0 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 6);
            x_in1 = __shfl_sync(FULL_MASK, input_to_warp, shifted_laneId | 7);

            asm(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
                " { %0, %1, %2, %3 },"
                " { %4, %5, %6, %7 },"
                " { %8, %9 },"
                " { %0, %1, %2, %3 };"
                : "+f"(z0), "+f"(z1), "+f"(z2), "+f"(z3)
                : "r"(w02), "r"(w12), "r"(w03), "r"(w13),
                  "r"(x_in0), "r"(x_in1)
            );
            }
        }

        // we produced 16 outputs, so only 16 threads
        if ((laneId & 1) == 0) {
            atomicAdd(output + (iin << 4) + (laneId >> 1), (laneId & 2) ? z2 : z0);
        }

        // if ((laneId & 3) == 0) {
        //     sum_scratch[warpId + ((laneId >> 1) + 0) * 32] = z0;
        //     sum_scratch[warpId + ((laneId >> 1) + 1) * 32] = z2;
        // }
        // __syncthreads();

        // // load and sum
        // if (warpId < 16) {
        //     float acc = sum_scratch[laneId + warpId*32];
        //     for (int offset = 16; offset > 0; offset /= 2) {
        //         acc += __shfl_down_sync(FULL_MASK, acc, offset);
        //     }
        //     if (laneId == 0) {
        //         output[(iin << 4) + warpId] = acc;
        //     }
        // }
    }
}


__host__ extern torch::Tensor decode_matvec_e8p(
    torch::Tensor x,
    torch::Tensor weights_compressed,
    torch::Tensor codebook_abs
) {

    CHECK_INPUT(x);
    CHECK_INPUT(weights_compressed);
    CHECK_INPUT(codebook_abs);

    TORCH_CHECK(x.dim() == 1);
    TORCH_CHECK(weights_compressed.dim() == 4);
    TORCH_CHECK(weights_compressed.size(3) == 4);
    TORCH_CHECK(weights_compressed.size(2) == 8);
    TORCH_CHECK(codebook_abs.dim() == 1);
    TORCH_CHECK(x.scalar_type() == torch::kFloat16);
    TORCH_CHECK(weights_compressed.scalar_type() == torch::kInt64);
    TORCH_CHECK(codebook_abs.scalar_type() == torch::kInt32);
    TORCH_CHECK(x.size(-1) == weights_compressed.size(1) << 6);
    TORCH_CHECK(codebook_abs.size(-1) == 256);

    int64_t N = weights_compressed.size(0) * 16;
    int64_t K = x.size(-1);

    TORCH_CHECK(K % 64 == 0, "K is not divisible by 64");
    TORCH_CHECK(N % 16 == 0, "N is not divisible by 16");

    TORCH_CHECK(K < 65536, "K is not too large");
    TORCH_CHECK(N < 65536, "N is not too large");

    at::DeviceGuard guard(x.device());
    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kFloat32)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{N}, options);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, x.get_device());
    int64_t grid_size = static_cast<int64_t>(deviceProp.multiProcessorCount);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    const dim3 block_size(32,32);

    decode_matvec_e8p_kernel<<<grid_size, block_size, 0, stream>>>(
        output.data_ptr<float>(),
        (const uint2*)x.data_ptr<c10::Half>(),
        (const uint2*)weights_compressed.data_ptr<int64_t>(),
        (const uint32_t*)codebook_abs.data_ptr<int32_t>(),
        N,
        K);
    
    gpuErrchk(hipPeekAtLastError());

    return output;
}



__global__ static void
test_tc_kernel(float *__restrict__ output) {
    int laneId = threadIdx.x;

    uint32_t w0 = (laneId == 0) ? 0x3C003C00 : 0x00000000;
    uint32_t w1 = 0x00000000;
    uint32_t w2 = 0x00000000;
    uint32_t w3 = 0x00000000;

    uint32_t x0 = (laneId == 0) ? 0x3C003C00 : 0x00000000;
    uint32_t x1 = 0x00000000;

    float z0 = 0.0;
    float z1 = 0.0;
    float z2 = 0.0;
    float z3 = 0.0;

    asm(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
        " { %0, %1, %2, %3 },"
        " { %4, %5, %6, %7 },"
        " { %8, %9 },"
        " { %0, %1, %2, %3 };"
        : "+f"(z0), "+f"(z1), "+f"(z2), "+f"(z3)
        : "r"(w0), "r"(w1), "r"(w2), "r"(w3),
          "r"(x0), "r"(x1)
    );

    output[laneId*4 + 0] = z0;
    output[laneId*4 + 1] = z1;
    output[laneId*4 + 2] = z2;
    output[laneId*4 + 3] = z3;
}

__host__ extern torch::Tensor test_tc() {

    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kFloat32)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{32*4}, options);

    test_tc_kernel<<<1, 32>>>(output.data_ptr<float>());
    
    gpuErrchk(hipPeekAtLastError());

    return output;
}




__global__ static void
test_codebook_expand_kernel(uint32_t *__restrict__ output, const uint32_t *__restrict__ codebook_abs) {
    uint32_t a = threadIdx.x;
    uint32_t b = 0;

    for (int i = 0; i < 8; i++) {
        b |= (((blockIdx.x >> i) & 1) << (4*i));
    }

    uint32_t s = b;
    s = s ^ (s >> 4);
    s = s ^ (s >> 8);
    s = s ^ (s >> 16);
    uint32_t sb = (s & 15);
    s = b ^ sb;
    sb = sb | (sb << 16);

    uint32_t x = codebook_abs[(a >> 0) & 255];
    x = x ^ ((s & 0x11111111) * 14);

    uint32_t o = BASE_OFFSET | ((sb & 0x00010001) << 4);

    uint32_t w0 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
    uint32_t w1 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
    uint32_t w2 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
    uint32_t w3 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

    output[blockIdx.x*256*4 + threadIdx.x*4 + 0] = w0;
    output[blockIdx.x*256*4 + threadIdx.x*4 + 1] = w1;
    output[blockIdx.x*256*4 + threadIdx.x*4 + 2] = w2;
    output[blockIdx.x*256*4 + threadIdx.x*4 + 3] = w3;
}

__host__ extern torch::Tensor test_codebook_expand(torch::Tensor codebook_abs) {

    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kFloat16)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{256*256,8}, options);

    test_codebook_expand_kernel<<<256, 256>>>((uint32_t*)output.data_ptr<c10::Half>(), (const uint32_t*)codebook_abs.data_ptr<int32_t>());
    
    gpuErrchk(hipPeekAtLastError());

    return output;
}




__global__ static void
// __launch_bounds__(1024, 1024)
decompress_packed_e8p_kernel(
    uint32_t *__restrict__ output,
    const uint2 *__restrict__ weights_compressed,
    const uint32_t *__restrict__ codebook_abs,
    int N,
    int K
) {
    int warpId = threadIdx.y;
    int laneId = threadIdx.x;

    for (int iin = blockIdx.x; iin < (N >> 4); iin += gridDim.x) {

        for (int iik = warpId; iik < (K >> 6); iik += 32) {
            uint2 w_compr = weights_compressed[laneId + 32*iik + (K >> 1)*iin];
            uint32_t a = w_compr.x;
            uint32_t b = w_compr.y;

            uint32_t s = b;
            s = s ^ (s >> 4);
            s = s ^ (s >> 8);
            s = s ^ (s >> 16);
            uint32_t sb = (s & 15);
            s = b ^ sb;
            sb = sb | (sb << 16);

            /// BLOCK 01
            {
            uint32_t x = codebook_abs[(a >> 0) & 255];
            x = x ^ ((s & 0x11111111) * 14);

            uint32_t o = BASE_OFFSET | ((sb & 0x00010001) << 4);

            uint32_t w00 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w01 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w02 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w03 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            x = codebook_abs[(a >> 8) & 255];
            x = x ^ ((s & 0x22222222) * 7);

            o = BASE_OFFSET | ((sb & 0x00020002) << 3);
            
            uint32_t w10 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w11 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w12 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w13 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 0] = w00;
            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 1] = w01;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 0] = w10;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 1] = w11;

            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 2] = w02;
            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 3] = w03;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 2] = w12;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 0*4 + ((laneId & 3) << 3) + 3] = w13;

            }
            /// BLOCK 23 
            {
            uint32_t x = codebook_abs[(a >> 16) & 255];
            s = s >> 2;
            x = x ^ ((s & 0x11111111) * 14);

            uint32_t o = BASE_OFFSET | ((sb & 0x00040004) << 2);
            
            uint32_t w00 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w01 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w02 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w03 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            x = codebook_abs[(a >> 24) & 255];
            x = x ^ ((s & 0x22222222) * 7);

            o = BASE_OFFSET | ((sb & 0x00080008) << 1); 

            uint32_t w10 = add_as_half2(mask_lop3(x << 4, XMASK, WMASK), o);
            uint32_t w11 = add_as_half2(mask_lop3(x << 0, XMASK, WMASK), o);
            uint32_t w12 = add_as_half2(mask_lop3(x >> 4, XMASK, WMASK), o);
            uint32_t w13 = add_as_half2(mask_lop3(x >> 8, XMASK, WMASK), o);

            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 0] = w00;
            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 1] = w01;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 0] = w10;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 1] = w11;

            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 2] = w02;
            output[iin*8*K + (laneId >> 2)*K + 0 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 3] = w03;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 2] = w12;
            output[iin*8*K + (laneId >> 2)*K + 1 * (K >> 1) + iik*32 + 1*4 + ((laneId & 3) << 3) + 3] = w13;
            }
        }
    }
}


__host__ extern torch::Tensor decompress_packed_e8p(
    torch::Tensor weights_compressed,
    torch::Tensor codebook_abs
) {
    CHECK_INPUT(weights_compressed);
    CHECK_INPUT(codebook_abs);

    TORCH_CHECK(weights_compressed.dim() == 4);
    TORCH_CHECK(weights_compressed.size(3) == 4);
    TORCH_CHECK(weights_compressed.size(2) == 8);
    TORCH_CHECK(codebook_abs.dim() == 1);
    TORCH_CHECK(weights_compressed.scalar_type() == torch::kInt64);
    TORCH_CHECK(codebook_abs.scalar_type() == torch::kInt32);
    TORCH_CHECK(codebook_abs.size(-1) == 256);

    int64_t N = weights_compressed.size(0) * 16;
    int64_t K = weights_compressed.size(1) << 6;

    TORCH_CHECK(K % 64 == 0, "K is not divisible by 64");
    TORCH_CHECK(N % 16 == 0, "N is not divisible by 16");

    TORCH_CHECK(K < 65536, "K is not too large");
    TORCH_CHECK(N < 65536, "N is not too large");

    at::DeviceGuard guard(codebook_abs.device());
    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kFloat16)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{N,K}, options);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, weights_compressed.get_device());
    int64_t grid_size = static_cast<int64_t>(deviceProp.multiProcessorCount);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    const dim3 block_size(32,32);

    decompress_packed_e8p_kernel<<<grid_size, block_size, 0, stream>>>(
        (uint32_t*)output.data_ptr<c10::Half>(),
        (const uint2*)weights_compressed.data_ptr<int64_t>(),
        (const uint32_t*)codebook_abs.data_ptr<int32_t>(),
        N,
        K);
    
    gpuErrchk(hipPeekAtLastError());

    return output;
}